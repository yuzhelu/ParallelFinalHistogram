#include "hip/hip_runtime.h"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
using namespace cv;
using namespace std;
#include <sys/time.h>
__global__ void histogramKernel(unsigned int *_hist, const unsigned char *_src, int _rows, int _cols)
{
	// get thread idx from built-in variables
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	// boundary check
	if (x > _cols - 1 || y > _rows - 1) return;

	int bin = _src[y * _cols + x];
	atomicInc(&_hist[bin], UINT_MAX);
}

void createHistogramGPU(unsigned int *_hist, Mat *_src)
{
	int cols = _src->cols;
	int rows = _src->rows;
	int dSize = cols * rows * sizeof(uchar);

	uchar *d_src;
	unsigned int *d_hist;

	// memory allocation
	hipMalloc((void **)& d_src, dSize);
	hipMalloc((void **)& d_hist, 256 * sizeof(unsigned int));

	// histogram initialization
	hipMemset(d_hist, 0, 256 * sizeof(unsigned int));

	// copy image to gpu
	hipMemcpy(d_src, _src->data, dSize, hipMemcpyHostToDevice);

	// calculate kernel threads, blocks size
	dim3 threads(16, 16);
	dim3 blocks((cols + 15) / 16, (rows + 15) / 16);

	// kernel call
	histogramKernel << <blocks, threads >> > (d_hist, d_src, rows, cols);

	// copy histogram to host memory
	hipMemcpy(_hist, d_hist, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

	// memory release
	hipFree(d_src);
	hipFree(d_hist);
}


int main()
{
	// Read image
	char pictures [11][10] = {"240.jpg", "240.jpg", "360.png", "480.jfif", "720.jpg", "1080.jpg", "4k.jpg", "8k.jpg", "16k.jpg" , "24k.jpg", "30k.jpg"};
	
	for(int i = 0; i < sizeof(pictures)/sizeof(pictures[0]); i++)
	{
        	cout<< "Pictures: " << *(pictures + i) << endl;
		Mat src = imread(*(pictures + i), IMREAD_COLOR);
		if (src.empty()) {
			cout << "Could not open or find the image" << endl;
			return -1;
		}
		// convert to gray 
		cvtColor(src, src, COLOR_BGR2GRAY);

		cout << "number of pixels on column: " << src.cols << endl;
		cout << "number of pixels on row: " << src.rows << endl;
		cout << "There is " << src.cols * src.rows << " number of pixels in the image" << endl;
		// create histogram variable
		unsigned int histogram[256];

		//unsigned int histogramGPU[256];
		//unsigned int histogramInvalidGPU[256];

		struct timeval tvalBefore, tvalAfter;  // removed comma
		gettimeofday (&tvalBefore, NULL);
		createHistogramGPU(histogram, &src);

   		gettimeofday (&tvalAfter, NULL);
 	   	// Changed format to long int (%ld), changed time calculation
		printf("Time in microseconds: %ld microseconds\n",
    	        ((tvalAfter.tv_sec - tvalBefore.tv_sec)*1000000L
   	 	       +tvalAfter.tv_usec) - tvalBefore.tv_usec
   		       ); // Added semicolon
	

	}	
	std::cout << "Press enter to exit the program" << endl;
	std::cin.ignore();
	return 0;
}

